#include "hip/hip_runtime.h"
#include<cstdio>
#include<hip/hip_runtime.h>

__global__ void VecAdd(float* A, float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

const int MAXN = 1024*1024;

int main() {
    size_t size = MAXN * sizeof(float);
    float *a = (float*)malloc(size);
    float *b = (float*)malloc(size);
    float *c = (float*)malloc(size);
    for (int i = 0; i < MAXN; ++i) a[i] = i;
    for (int i = 0; i < MAXN; ++i) b[i] = i;
    printf("size: %ld\n", size);

    float* dA;
    hipMalloc((void **)&dA, size);
    hipError_t e2 = hipGetLastError();
    if (e2 != hipSuccess) {
        printf("ERROR: %d %s\n", e2, hipGetErrorString(e2));
    }

    hipMemcpy(dA, a, size, hipMemcpyHostToDevice);
    float* dB;
    hipMalloc((void **)&dB, size);
    hipMemcpy(dB, b, size, hipMemcpyHostToDevice);
    float* dC;
    hipMalloc((void **)&dC, size);

    clock_t start_d = clock();
    hipDeviceSynchronize();
    VecAdd<<<(MAXN+255)/256, 256>>>(dA, dB, dC, MAXN);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %d %s\n", error, hipGetErrorString(error));
    }
    clock_t end_d = clock();
    printf("clock %lf\n", (double)(end_d-start_d)/CLOCKS_PER_SEC);

    hipMemcpy(c, dC, size, hipMemcpyDeviceToHost);
    printf("%f %f %f\n", a[1024], b[1024], c[1024]);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}
