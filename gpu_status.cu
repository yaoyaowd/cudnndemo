#include <stdio.h> 

struct hipFuncAttributes funcAttrib;

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(KERNEL));
    printf("%s numRegs=%d\n",KERNELNAME,funcAttrib.numRegs);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        int support = 0;
        hipDeviceGetAttribute(&support, hipDeviceAttributeStreamPrioritiesSupported, 0);
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Priorities support: %d\n", support);
        if (support) {
          int leastP = 0, greatP = 0;
          hipDeviceGetStreamPriorityRange(&leastP, &greatP);
          printf("leastPriority %d, greatestPriority %d\n", leastP, greatP);
        }
        printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 
                2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        // printf("%d\n", prop.maxThreadsDim);
        printf("%d\n", prop.maxThreadsPerBlock);
    }
    return 0;
}
