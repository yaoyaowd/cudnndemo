#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

int main(int argc, const char* argv[]) {
  int gpu_id = 0;
  std::cerr << "GPU: " << gpu_id << std::endl;

  std::vector<short> image(32 * 300 * 2944);
  int rows = 300;
  int cols = 2944;

  hipSetDevice(gpu_id);

  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*dataType=*/HIPDNN_DATA_HALF,
                                        /*batch_size=*/1,
                                        /*channels=*/32,
                                        /*image_height=*/rows,
                                        /*image_width=*/cols));

  hipdnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                        /*dataType=*/HIPDNN_DATA_HALF,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*out_channels=*/32,
                                        /*in_channels=*/32,
                                        /*kernel_height=*/3,
                                        /*kernel_width=*/3));

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                             /*pad_height=*/1,
                                             /*pad_width=*/1,
                                             /*vertical_stride=*/1,
                                             /*horizontal_stride=*/1,
                                             /*dilation_height=*/1,
                                             /*dilation_width=*/1,
                                             /*mode=*/HIPDNN_CROSS_CORRELATION,
                                             /*computeType=*/HIPDNN_DATA_HALF));
  checkCUDNN(hipdnnSetConvolutionMathType(convolution_descriptor, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));

  int batch_size{0}, channels{0}, height{0}, width{0};
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                   input_descriptor,
                                                   kernel_descriptor,
                                                   &batch_size,
                                                   &channels,
                                                   &height,
                                                   &width));

  std::cerr << "Output Image: " << channels << " x " << height << " x " << width
            << std::endl;

  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*dataType=*/HIPDNN_DATA_HALF,
                                        /*batch_size=*/1,
                                        /*channels=*/32,
                                        /*image_height=*/rows,
                                        /*image_width=*/cols));

  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  checkCUDNN(
      hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                          input_descriptor,
                                          kernel_descriptor,
                                          convolution_descriptor,
                                          output_descriptor,
                                          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                          /*memoryLimitInBytes=*/0,
                                          &convolution_algorithm));
  printf("%d\n", convolution_algorithm);
  //convolution_algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED;
  //printf("%d\n", convolution_algorithm);

  size_t workspace_bytes{0};
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     convolution_descriptor,
                                                     output_descriptor,
                                                     convolution_algorithm,
                                                     &workspace_bytes));
  std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB"
            << std::endl;
  assert(workspace_bytes > 0);

  void* d_workspace{nullptr};
  hipMalloc(&d_workspace, workspace_bytes);

  int image_bytes = batch_size * 32 * height * width * sizeof(short);
  int output_bytes = batch_size * 32 * height * width * sizeof(short);

  short* d_input{nullptr};
  hipMalloc(&d_input, image_bytes);
  hipMemcpy(d_input, image.data(), image_bytes, hipMemcpyHostToDevice);

  short* d_output{nullptr};
  hipMalloc(&d_output, output_bytes);
  hipMemset(d_output, 0, output_bytes);

  // clang-format off
  const short kernel_template[3][3] = {
    {1, 1, 1},
    {1, -8, 1},
    {1, 1, 1}
  };
  // clang-format on

  short h_kernel[32][3][3][3];
  for (int kernel = 0; kernel < 32; ++kernel) {
    for (int channel = 0; channel < 32; ++channel) {
      for (int row = 0; row < 3; ++row) {
        for (int column = 0; column < 3; ++column) {
          h_kernel[kernel][channel][row][column] = kernel_template[row][column];
        }
      }
    }
  }

  short* d_kernel{nullptr};
  hipMalloc(&d_kernel, sizeof(h_kernel));
  hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

  const float alpha = 1.0f, beta = 0.0f;

  hipEvent_t start_event, stop_event;
  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);
  checkCUDNN(hipdnnConvolutionForward(cudnn,
                                     &alpha,
                                     input_descriptor,
                                     d_input,
                                     kernel_descriptor,
                                     d_kernel,
                                     convolution_descriptor,
                                     convolution_algorithm,
                                     d_workspace,
                                     workspace_bytes,
                                     &beta,
                                     output_descriptor,
                                     d_output));
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  float time;
  hipEventElapsedTime(&time, start_event, stop_event);
  printf("conv took %f ms\n", time);
/*
  short* h_output = new short[output_bytes];
  hipMemcpy(h_output, d_output, output_bytes, hipMemcpyDeviceToHost);

  delete[] h_output;
  */
  hipFree(d_kernel);
  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_workspace);

  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

  hipdnnDestroy(cudnn);
}
