#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

typedef float DataType;

int main(int argc, const char* argv[]) {
  hipSetDevice(0);

  int in = 1, ic = 32, irow = 256, icol = 2944;
  int on = 1, oc = 32, orow = 128, ocol = 1472;
  hipdnnDataType_t cudnn_type = sizeof(DataType) == 4 ? HIPDNN_DATA_FLOAT : HIPDNN_DATA_HALF;

  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*dataType=*/cudnn_type,
                                        /*batch_size=*/in,
                                        /*channels=*/ic,
                                        /*image_height=*/irow,
                                        /*image_width=*/icol));

  hipdnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                        /*dataType=*/cudnn_type,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*out_channels=*/oc,
                                        /*in_channels=*/ic,
                                        /*kernel_height=*/3,
                                        /*kernel_width=*/3));

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                             /*pad_height=*/1,
                                             /*pad_width=*/1,
                                             /*vertical_stride=*/2,
                                             /*horizontal_stride=*/2,
                                             /*dilation_height=*/1,
                                             /*dilation_width=*/1,
                                             /*mode=*/HIPDNN_CROSS_CORRELATION,
                                             /*computeType=*/cudnn_type));
  checkCUDNN(hipdnnSetConvolutionMathType(convolution_descriptor, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));

  int batch_size{0}, channels{0}, height{0}, width{0};
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                   input_descriptor,
                                                   kernel_descriptor,
                                                   &batch_size,
                                                   &channels,
                                                   &height,
                                                   &width));

  std::cerr << "Output Image: " << channels << " x " << height << " x " << width
            << std::endl;

  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*dataType=*/cudnn_type,
                                        /*batch_size=*/on,
                                        /*channels=*/oc,
                                        /*image_height=*/orow,
                                        /*image_width=*/ocol));

  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  checkCUDNN(
      hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                          input_descriptor,
                                          kernel_descriptor,
                                          convolution_descriptor,
                                          output_descriptor,
                                          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                          /*memoryLimitInBytes=*/2 * 1024 * 1024 * 1024,
                                          &convolution_algorithm));
  printf("%d\n", convolution_algorithm);
  //convolution_algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED;
  //printf("%d\n", convolution_algorithm);

  size_t workspace_bytes{0};
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     convolution_descriptor,
                                                     output_descriptor,
                                                     convolution_algorithm,
                                                     &workspace_bytes));
  std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB"
            << std::endl;
  assert(workspace_bytes > 0);

  void* d_workspace{nullptr};
  hipMalloc(&d_workspace, workspace_bytes);

  int image_bytes = in * ic * irow * icol * sizeof(DataType);
  int output_bytes = on * oc * orow * ocol * sizeof(DataType);

  short* d_input{nullptr};
  hipMalloc(&d_input, image_bytes);
  hipMemset(d_input, 1, image_bytes);

  short* d_output{nullptr};
  hipMalloc(&d_output, output_bytes);
  hipMemset(d_output, 0, output_bytes);

  // clang-format off
  const short kernel_template[3][3] = {
    {1, 1, 1},
    {1, -8, 1},
    {1, 1, 1}
  };
  // clang-format on

  short h_kernel[256][256][3][3];
  for (int kernel = 0; kernel < ic; ++kernel) {
    for (int channel = 0; channel < oc; ++channel) {
      for (int row = 0; row < 3; ++row) {
        for (int column = 0; column < 3; ++column) {
          h_kernel[kernel][channel][row][column] = kernel_template[row][column];
        }
      }
    }
  }

  short* d_kernel{nullptr};
  hipMalloc(&d_kernel, sizeof(h_kernel));
  hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

  const float alpha = 1.0f, beta = 0.0f;

  hipEvent_t start_event, stop_event;
  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);
  checkCUDNN(hipdnnConvolutionForward(cudnn,
                                     &alpha,
                                     input_descriptor,
                                     d_input,
                                     kernel_descriptor,
                                     d_kernel,
                                     convolution_descriptor,
                                     convolution_algorithm,
                                     d_workspace,
                                     workspace_bytes,
                                     &beta,
                                     output_descriptor,
                                     d_output));
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  float time;
  hipEventElapsedTime(&time, start_event, stop_event);
  printf("conv took %f ms\n", time);
/*
  short* h_output = new short[output_bytes];
  hipMemcpy(h_output, d_output, output_bytes, hipMemcpyDeviceToHost);

  delete[] h_output;
  */
  hipFree(d_kernel);
  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_workspace);

  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

  hipdnnDestroy(cudnn);
}
